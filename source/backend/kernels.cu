#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"

__global__ static void poisson_logpmf_kernel(
    const int *k, const double *r, double *out, const int size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size)
  {
    return;
  }

  out[i] = k[i] * logf(r[i]) - r[i] - lgammaf(k[i]+1);
}

// k is data in host RAM
// r is data in host RAM
// out is data in host RAM
void poisson_logpmf_np_and_np_to_np(
    const int *k, const double *r, double *out, const int size)
{
  //int stream_size = size / NUM_STREAMS;

  //hipStream_t streams[NUM_STREAMS];
  //for (int i = 0; i < NUM_STREAMS; i++)
  //{
    //cuda_errchk(hipStreamCreate(&streams[i]));
  //}

  int *k_d;
  double *r_d, *out_d;
  hipMalloc(&k_d, sizeof(int)*size);
  hipMalloc(&r_d, sizeof(double)*size);
  hipMalloc(&out_d, sizeof(double)*size);

  //for (int i = 0; i < NUM_STREAMS; i++)
  //{
    //int offset = i * stream_size;
    //int chunk_size = (i < NUM_STREAMS-1) ? stream_size : size-offset;
  int num_blocks = size / THREAD_BLOCK_SIZE + (size % THREAD_BLOCK_SIZE > 0);

  cuda_errchk(hipMemcpy(k_d, k, 
        sizeof(int)*size, hipMemcpyHostToDevice));
  cuda_errchk(hipMemcpy(r_d, r, 
        sizeof(double)*size, hipMemcpyHostToDevice));

  poisson_logpmf_kernel<<<num_blocks, THREAD_BLOCK_SIZE>>>(
      k_d, r_d, out_d, size);

  cuda_errchk(hipMemcpy(out, out_d, 
        sizeof(double)*size, hipMemcpyDeviceToHost));
  //}

  cuda_errchk(hipDeviceSynchronize());
  cuda_errchk(hipFree(k_d));
  cuda_errchk(hipFree(r_d));
  cuda_errchk(hipFree(out_d));
}

// k is data in GPU global memory
// r is data in host RAM
// out is data in GPU global memory
void poisson_logpmf_cp_and_np_to_cp(
    const int *k, const double *r, double *out, const int size)
{
  //int stream_size = size / NUM_STREAMS;

  //hipStream_t streams[NUM_STREAMS];
  //for (int i = 0; i < NUM_STREAMS; i++)
  //{
    //cuda_errchk(hipStreamCreate(&streams[i]));
  //}

  double *r_d;
  hipMalloc(&r_d, sizeof(double)*size);

  //for (int i = 0; i < NUM_STREAMS; i++)
  //{
    //int offset = i * stream_size;
    //int chunk_size = (i < NUM_STREAMS-1) ? stream_size : size-offset;
  int num_blocks = size / THREAD_BLOCK_SIZE + (size % THREAD_BLOCK_SIZE > 0);

  cuda_errchk(hipMemcpy(r_d, r, 
        sizeof(double)*size, hipMemcpyHostToDevice));

  poisson_logpmf_kernel<<<num_blocks, THREAD_BLOCK_SIZE>>>(
      k, r_d, out, size);
  //}

  cuda_errchk(hipDeviceSynchronize());
  cuda_errchk(hipFree(r_d));
}

// k is data in GPU global memory
// r is data in GPU global memory
// out is data in GPU global memory
void poisson_logpmf_cp_and_cp_to_cp(
    const int *k, const double *r, double *out, const int size)
{
  int num_blocks = size / THREAD_BLOCK_SIZE + (size % THREAD_BLOCK_SIZE > 0);
  poisson_logpmf_kernel<<<num_blocks, THREAD_BLOCK_SIZE>>>(
      k, r, out, size);
  cuda_errchk(hipDeviceSynchronize());
}
